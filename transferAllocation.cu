#include <hip/hip_runtime.h>
#include "cell.h"
#include "LogSystem/FileLogger.hpp"

#define HANDLE_CUERROR(call) {										             \
    hipError_t err = call;												         \
    if(err != hipSuccess) {											         \
        *((logging::FileLogger*)Log) << _ERROR_                                  \
                       << (std::string("CUDA error in file '") +                 \
                           std::string(__FILE__) +  std::string("' in line ") +  \
                           std::to_string(__LINE__) + std::string(": ") +        \
                           std::string(hipGetErrorString(err))).c_str();        \
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	         \
            __FILE__, __LINE__, hipGetErrorString(err));				         \
        exit(1);														         \
    }																	         \
} while (0)

extern "C" void cu_AllocateGpuMemory(void** ptr, int size, void* Log)
{
    HANDLE_CUERROR( hipMalloc( ptr, size ) );
}

extern "C" void cu_AllocateHostPinnedMemory(void** ptr, int size, void* Log)
{
    HANDLE_CUERROR( hipHostAlloc(ptr, size, hipHostMallocDefault) );
}

extern "C" void cu_FreeGpuMemory(Cell* ptr, void* Log)
{
    HANDLE_CUERROR(hipFree(ptr));
}

extern "C" void cu_FreeHostPinnedMemory(Cell* ptr, void* Log)
{
    HANDLE_CUERROR(hipHostFree(ptr));
}

extern "C" void cu_loadDataToGpu(Cell* dev, Cell* host, int size, void* Log)
{
    HANDLE_CUERROR( hipMemcpy( dev, host, size, hipMemcpyHostToDevice ) );
}

extern "C" void cu_loadDataToHost(Cell* host, Cell* dev, int size, void* Log)
{
    HANDLE_CUERROR( hipMemcpy( host, dev, size, hipMemcpyDeviceToHost ) );
}
