#include <hip/hip_runtime.h>
#include "cell.h"
#include "LogSystem/FileLogger.hpp"
#include "cu_gpuProperties.h"

extern "C" void cu_AllocateHostPinnedMemory(void** ptr, int size, void* Log)
{
    HANDLE_CUERROR( hipHostAlloc(ptr, size, hipHostMallocDefault) );
}

extern "C" void cu_FreeHostPinnedMemory(Cell* ptr, void* Log)
{
    HANDLE_CUERROR(hipHostFree(ptr));
}

extern "C" void cu_AllocateFieldMemory(void* prop, int size)
{
    cu_gpuProperties* gpu = (cu_gpuProperties*) prop;
    logging::FileLogger* Log = gpu->Log;
    HANDLE_CUERROR( hipMalloc( (void**)&gpu->m_Field, size ) );
}

extern "C" void cu_AllocateHaloMemory(void* prop, int size)
{
    cu_gpuProperties* gpu = (cu_gpuProperties*) prop;
    logging::FileLogger* Log = gpu->Log;
    HANDLE_CUERROR( hipMalloc( (void**)&gpu->m_halo, size ) );
}

extern "C" void cu_loadFieldData(void* prop, Cell* host, int size, int type)
{ // type = { cu_loadFromDeviceToHost, cu_loadFromHostToDevice }
    cu_gpuProperties* gpu = (cu_gpuProperties*) prop;
    logging::FileLogger* Log = gpu->Log;
    if(type == cu_loadFromDeviceToHost) {
        HANDLE_CUERROR( hipMemcpy( host, gpu->m_Field, size, hipMemcpyDeviceToHost ) );
    } else if(type == cu_loadFromHostToDevice) {
        HANDLE_CUERROR( hipMemcpy( gpu->m_Field, host, size, hipMemcpyHostToDevice ) );
    } else {
        *Log << _WARNING_ << "Wrong 'type' in 'cu_loadFieldData'. Nothing will be done.";
    }
}

extern "C" void cu_loadHaloData(void* prop, Cell* host, int size, int type)
{ // type = { cu_loadFromDeviceToHost, cu_loadFromHostToDevice }
    cu_gpuProperties* gpu = (cu_gpuProperties*) prop;
    logging::FileLogger* Log = gpu->Log;
    if(type == cu_loadFromDeviceToHost) {
        HANDLE_CUERROR( hipMemcpy( host, gpu->m_halo, size, hipMemcpyDeviceToHost ) );
    } else if(type == cu_loadFromHostToDevice) {
        HANDLE_CUERROR( hipMemcpy( gpu->m_halo, host, size, hipMemcpyHostToDevice ) );
    } else {
        *Log << _WARNING_ << "Wrong 'type' in 'cu_loadHaloData'. Nothing will be done.";
    }
}
