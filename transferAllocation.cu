#include <hip/hip_runtime.h>
#include "cell.h"
#include "LogSystem/FileLogger.hpp"

#define HANDLE_CUERROR(call) {										             \
    hipError_t err = call;												         \
    if(err != hipSuccess) {											         \
        *((logging::FileLogger*)Log) << _ERROR_                                  \
                       << (std::string("CUDA error in file '") +                 \
                           std::string(__FILE__) +  std::string("' in line ") +  \
                           std::to_string(__LINE__) + std::string(": ") +        \
                           std::string(hipGetErrorString(err))).c_str();        \
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	         \
            __FILE__, __LINE__, hipGetErrorString(err));				         \
        exit(1);														         \
    }																	         \
} while (0)

extern "C" void AllocateGpuMemory(void** ptr, int size, void* Log)
{
    HANDLE_CUERROR( hipMalloc( ptr, size ) );
}

extern "C" void AllocateHostPinnedMemory(void** ptr, int size, void* Log)
{
    HANDLE_CUERROR( hipHostAlloc(ptr, size, hipHostMallocDefault) );
}

extern "C" void FreeGpuMemory(Cell* ptr, void* Log)
{
    HANDLE_CUERROR(hipFree(ptr));
}

extern "C" void FreeHostPinnedMemory(Cell* ptr, void* Log)
{
    HANDLE_CUERROR(hipHostFree(ptr));
}

extern "C" void loadDataToGpu(Cell* dev, Cell* host, int size, void* Log)
{
    HANDLE_CUERROR( hipMemcpy( dev, host, size, hipMemcpyHostToDevice ) );
}

extern "C" void loadDataToHost(Cell* host, Cell* dev, int size, void* Log)
{
    HANDLE_CUERROR( hipMemcpy( host, dev, size, hipMemcpyDeviceToHost ) );
}
